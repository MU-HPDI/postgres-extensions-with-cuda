#include "hip/hip_runtime.h"
#include "cuda_kernel.hpp"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/iterator_facade.h>
#define BLOCK_SIZE 1024

void cuda_wrapper_vector_addition(int *x, int *y, int *result, int n){
    // Set device to get it warmed up before we need it
    hipSetDevice(0);

    int threads_per_block = 10;
    int no_of_blocks = n / threads_per_block;


    int *x_d, *y_d, *result_d;
    hipMalloc((void **)&x_d, n*sizeof(int));
    hipMalloc((void **)&y_d, n*sizeof(int));
    hipMalloc((void **)&result_d, n*sizeof(int));
    
    hipMemcpy(x_d, x, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, n*sizeof(int), hipMemcpyHostToDevice);
    //  === CUDA ===
    vector_add_cuda<<<no_of_blocks,threads_per_block>>>(x_d, y_d, result_d, n);

    hipMemcpy(result, result_d, n*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(x_d); hipFree(y_d); hipFree(result_d);
}


void cuda_find_max(const short int* data, short int *output, int rows, int cols) 
{

    thrust::device_vector<short int> d_data(rows * cols);
    thrust::copy(data, data + (rows * cols), d_data.begin());

    short int *d_data_ptr = thrust::raw_pointer_cast(d_data.data());

    dim3 blockSize(BLOCK_SIZE, 1, 1);
    dim3 gridSize(ceil((float)cols / BLOCK_SIZE), rows, 1);

    thrust::device_vector<short int> d_result(rows);
    short int *d_result_ptr = thrust::raw_pointer_cast(d_result.data());

    max_kernel<<<gridSize, blockSize>>>(d_data_ptr, d_result_ptr, rows, cols);
    hipDeviceSynchronize();

    // get last CUDA error
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }

    thrust::host_vector<short int> h_result = d_result;

    // copy h_result to output
    for (int i = 0; i < rows; i++)
    {
        output[i] = h_result[i];
    }

}