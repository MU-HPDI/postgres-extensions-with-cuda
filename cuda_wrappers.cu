#include "cuda_kernel.hpp"

void cuda_wrapper_vector_addition(int *x, int *y, int *result, int n){
    // Set device to get it warmed up before we need it
    hipSetDevice(0);

    int threads_per_block = 10;
    int no_of_blocks = n / threads_per_block;


    int *x_d, *y_d, *result_d;
    hipMalloc((void **)&x_d, n*sizeof(int));
    hipMalloc((void **)&y_d, n*sizeof(int));
    hipMalloc((void **)&result_d, n*sizeof(int));
    
    hipMemcpy(x_d, x, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, n*sizeof(int), hipMemcpyHostToDevice);
    //  === CUDA ===
    vector_add_cuda<<<no_of_blocks,threads_per_block>>>(x_d, y_d, result_d, n);

    hipMemcpy(result, result_d, n*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(x_d); hipFree(y_d); hipFree(result_d);
}
